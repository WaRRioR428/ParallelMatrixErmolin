﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <cstdlib>

using namespace std;

const int matrixSize = 3300;

__global__ void matrixMult(const int* A, const int* B, int* C)
{
    int i = matrixSize * (blockDim.y * blockIdx.y + threadIdx.y);
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int sum = 0;

    for (int k = 0; k < matrixSize; k++)
        sum += A[i + k] * B[k * matrixSize + j];

    int ind = matrixSize * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    C[ind] = sum;
}

int main(int argc, char** argv) {

    size_t size = matrixSize * matrixSize * sizeof(int);

    int* a = (int*)malloc(size);
    int* b = (int*)malloc(size);
    int* c = (int*)malloc(size);

    cout << "Filling matrixes" << endl;
    cout << endl;

    for (int i = 0; i < matrixSize; i++) {
        for (int j = 0; j < matrixSize; j++) {
            a[i * matrixSize + j] = rand() % 100;
            b[i * matrixSize + j] = rand() % 100;
        }
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int* calcA = NULL;
    hipMalloc((void**)&calcA, size);

    int* calcB = NULL;
    hipMalloc((void**)&calcB, size);

    int* calcC = NULL;
    hipMalloc((void**)&calcC, size);

    hipMemcpy(calcA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(calcB, b, size, hipMemcpyHostToDevice);

    cout << "Starting calculations" << endl;
    cout << endl;

    int maxThreads = 4;

    for (int t = 1; t <= maxThreads; t++)
    {
        dim3 threadsPerBlock = dim3(t, t);
        dim3 blocksPerGrid = dim3(matrixSize / t, matrixSize / t);

        hipEventRecord(start, 0);
        matrixMult<<<blocksPerGrid, threadsPerBlock>>>(calcA, calcB, calcC);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float kernelTime;
        hipEventElapsedTime(&kernelTime, start, stop);
        printf("Threads per block: %d; Blocks per grid: %d; KernelTime: %d seconds\n", t, matrixSize / t, (int)(kernelTime / 1000));

        hipMemcpy(c, calcC, size, hipMemcpyDeviceToHost);
    }

    hipFree(calcA);
    hipFree(calcB);
    hipFree(calcC);
    free(a);
    free(b);
    free(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
